// Author : Yash Deshpande
// Date : 
// Tutor : Izzat El Hajj, AUB

#include <iostream>
#include <hip/hip_runtime.h>
#include <random>               // generate random numbers
#include <chrono>               // std::chrono namespace provides timer functions in C++, for CPU timing
#include <ratio>                // std::ratio provides easy conversions between metric units, for CPU timing

using std::chrono::duration;
using std::chrono::high_resolution_clock;

// prints all the elements of given array
void printArr(unsigned char *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        std::cout << int(arr[i]) << " ";
    }
    std::cout << std::endl;
}

int main(int argc, char **argv)
{
    hipDeviceSynchronize();

    // ######################### Allocate memory & initialize data #########################

    // Declare the host arrays

    // Generate Random Numbers
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);

    // Initialize the arrays


    // ######################### vector addition on CPU #########################

    high_resolution_clock::time_point start_cpu;
    high_resolution_clock::time_point end_cpu;
    duration<double, std::milli> duration_sec;

    //////////////////////////////////////
    start_cpu = high_resolution_clock::now(); // Get the starting timestamp
    
    end_cpu = high_resolution_clock::now(); // Get the ending timestamp
    //////////////////////////////////////

    duration_sec = std::chrono::duration_cast<duration<double, std::milli>>(end_cpu - start_cpu);
    std::cout << "Total CPU time: " << duration_sec.count() << "ms\n";
    

    // ######################### vector addition on GPU #########################
    hipEvent_t start_gpu;
    hipEvent_t stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    //////////////////////////////////////
    hipEventRecord(start_gpu);
    
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    //////////////////////////////////////

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start_gpu, stop_gpu);
    std::cout << "Total GPU time: " << ms << "ms\n";
    
    // free host memory
    

    return 0;
}