// Author : Yash Deshpande
// Date : 29-12-2024
// Tutor : Izzat El Hajj, AUB

#include <iostream>
#include <hip/hip_runtime.h>
#include <random> // generate random numbers
#include <chrono> // std::chrono namespace provides timer functions in C++, for CPU timing
#include <ratio>  // std::ratio provides easy conversions between metric units, for CPU timing

using std::chrono::duration;
using std::chrono::high_resolution_clock;

// Tile dimension : 32x32
// Block dimension : 32x32
#define TILE_DIM 32

// declare an arbitrary coarsening factor
#define COARSE_FACTOR 4

// prints all the elements of given array
void printArr(float *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}

// performs matrix multiplication on the CPU
void matmul_tiled_cpu(float *A, float *B, float *C, unsigned int N)
{
    // loop over every row tile
    for (int row_tile = 0; row_tile < N / TILE_DIM; row_tile++)
    {
        // loop over every col tile
        for (int col_tile = 0; col_tile < N / TILE_DIM; col_tile++)
        {
            // loop over every input tile
            for (int i_tile = 0; i_tile < N / TILE_DIM; i_tile++)
            {
                // loop over every row within the tile
                for (int row = row_tile * TILE_DIM; row < (row_tile + 1) * TILE_DIM; row++)
                {
                    // loop over every col withing the tile
                    for (int col = col_tile * TILE_DIM; col < (col_tile + 1) * TILE_DIM; col++)
                    {
                        float sum = 0.0f;
                        for (int i = i_tile * TILE_DIM; i < (i_tile + 1) * TILE_DIM; i++)
                        {
                            sum += A[row * N + i] * B[i * N + col];
                        }
                        if (i_tile == 0)
                        {
                            C[row * N + col] = sum;
                        }
                        else
                        {
                            C[row * N + col] += sum;
                        }
                    }
                }
            }
        }
    }
}

__global__ void matmul_tiled_kernel(float *A, float *B, float *C, unsigned int N)
{
    // declare arrays in shared memory
    __shared__ float A_s[TILE_DIM][TILE_DIM];
    __shared__ float B_s[TILE_DIM][TILE_DIM];

    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col_start = blockDim.x * blockIdx.x * COARSE_FACTOR + threadIdx.x;

    float sum[COARSE_FACTOR];
    for (unsigned int c = 0; c < COARSE_FACTOR; c++)
    {
        sum[c] = 0.0f;
    }

    // loop over each tile
    for (unsigned int tile = 0; tile < N / TILE_DIM; tile++)
    {
        // load tile to shared memory
        A_s[threadIdx.y][threadIdx.x] = A[row * N + tile * TILE_DIM + threadIdx.x];

        for (unsigned int c = 0; c < COARSE_FACTOR; c++)
        {
            unsigned int col = col_start + c*TILE_DIM;
            B_s[threadIdx.y][threadIdx.x] = B[(tile * TILE_DIM + threadIdx.y) * N + col];
            __syncthreads(); // Threads wait for each other to finish loading before computing

            // calculate partial sums with tile
            for (unsigned int i = 0; i < TILE_DIM; i++)
            {
                sum[c] += A_s[threadIdx.y][i] * B_s[i][threadIdx.x];
            }
            __syncthreads(); // Threads wait for each other to finish computing before loading
        }
    }

    for (unsigned int c = 0; c < COARSE_FACTOR; c++)
    {
        unsigned int col = col_start + c*TILE_DIM;
        C[row * N + col] = sum[c];
    }
}

void matmul_tiled_gpu(float *A, float *B, float *C, unsigned int N)
{
    // for timing purpose
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate memory on GPU
    float *A_d, *B_d, *C_d;
    hipMalloc((void **)&A_d, N * N * sizeof(float));
    hipMalloc((void **)&B_d, N * N * sizeof(float));
    hipMalloc((void **)&C_d, N * N * sizeof(float));

    // Copy data to GPU
    hipMemcpy(A_d, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Run Kernel
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x/COARSE_FACTOR, (N + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);

    //////////////////////////////////////
    hipEventRecord(start);
    matmul_tiled_kernel<<<numBlocks, numThreadsPerBlock>>>(A_d, B_d, C_d, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //////////////////////////////////////

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Total GPU Kernel time: " << ms << "ms\n";

    // Copy data from GPU
    hipMemcpy(C, C_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Deallocate memory on GPU
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(int argc, char **argv)
{
    hipDeviceSynchronize();

    // ######################### Allocate memory & initialize data #########################

    // Declare the host arrays
    unsigned int N = (argc > 1) ? atoi(argv[1]) : (1024);
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));

    // Generate Random Numbers
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-10.0, 10.0);

    // Initialize the arrays
    for (int i = 0; i < N * N; i++)
    {
        A[i] = dist(generator);
        B[i] = dist(generator);
    }
    // printArr(A, N * N);
    // printArr(B, N * N);

    // ######################### vector addition on CPU #########################

    high_resolution_clock::time_point start_cpu;
    high_resolution_clock::time_point end_cpu;
    duration<double, std::milli> duration_sec;

    //////////////////////////////////////
    start_cpu = high_resolution_clock::now(); // Get the starting timestamp
    matmul_tiled_cpu(A, B, C, N);
    end_cpu = high_resolution_clock::now(); // Get the ending timestamp
    //////////////////////////////////////

    duration_sec = std::chrono::duration_cast<duration<double, std::milli>>(end_cpu - start_cpu);
    std::cout << "Total CPU time: " << duration_sec.count() << "ms\n";

    // ######################### vector addition on GPU #########################
    hipEvent_t start_gpu;
    hipEvent_t stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    //////////////////////////////////////
    hipEventRecord(start_gpu);
    matmul_tiled_gpu(A, B, C, N);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    //////////////////////////////////////

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start_gpu, stop_gpu);
    std::cout << "Total GPU time: " << ms << "ms\n";

    // free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}