// Author : Yash Deshpande
// Date : 06-01-2024
// Tutor : Izzat El Hajj, AUB

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <random>       // generate random numbers
#include <chrono>       // std::chrono namespace provides timer functions in C++, for CPU timing
#include <ratio>        // std::ratio provides easy conversions between metric units, for CPU timing

using std::chrono::duration;
using std::chrono::high_resolution_clock;


// ######################## Run Commands #######################
// 1. Compile the program               : nvcc <file_name>.cu -o <binary_name>
// 2. Run the profiler                  : nvprof ./<binary_name> 
// 3. copy profiler content to a file   : nvprof -o <output_file_name> ./<binary_name> 
// 4. More info from profiler           : nvprof -m all -o <output_file_name> ./<binary_name> 
// 5. Run Nvidia Visual Profiler        : nvvp &  (does not work)
// NOTE : nvprof & nvvp have been replaced by Nsight now 
// #############################################################

// print all the elements of given array
void printArr(float *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}

// adds 2 float values
__device__ __host__ float f_add (float a, float b)
{
    return a + b;
}



// CUDA kernel to add 2 vectors
__global__ void vecadd_kernel(float *x, float *y, float *z, int N)
{
    // calculate global index
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        // z[i] = x[i] + y[i];
        z[i] = f_add(x[i], y[i]);
    }
}

// parallel vector addition on GPU
void vecadd_gpu(float *x, float *y, float *z, int N)
{
    // for timing purpose
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    

    // Allocate GPU Memory
    float *x_d, *y_d, *z_d;
    hipMalloc((void **)&x_d, N * sizeof(float));
    hipMalloc((void **)&y_d, N * sizeof(float));
    hipMalloc((void **)&z_d, N * sizeof(float));

    // Copy to the GPU
    hipMemcpy(x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Run the GPU Code
    int numThreadsPerBlock = 512;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;
    
    //////////////////////////////////////
    hipEventRecord(start);
    vecadd_kernel<<<numBlocks, numThreadsPerBlock>>>(x_d, y_d, z_d, N);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //////////////////////////////////////

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "Total GPU Kernel time: " << ms << "ms\n";
    

    // Copy from GPU
    hipMemcpy(z, z_d, N * sizeof(float), hipMemcpyDeviceToHost);

    // Deallocate GPU Memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}

int main(int argc, char **argv)
{
    hipDeviceSynchronize();

    // ######################### Allocate memory & initialize data #########################
    unsigned int N = (argc > 1) ? atoi(argv[1]) : (1 << 25);
    float *x = (float *)malloc(N * sizeof(float));
    float *y = (float *)malloc(N * sizeof(float));
    float *z = (float *)malloc(N * sizeof(float));

    // Generate Random Numbers b/w [10, 20]
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(10., 20.);

    for (int i = 0; i < N; i++)
    {
        x[i] = dist(generator);
        y[i] = dist(generator);
    }
    // printArr(x, N);
    // printArr(y, N);

    // ######################### vector addition on GPU #########################
    hipEvent_t start_gpu;
    hipEvent_t stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    //////////////////////////////////////
    hipEventRecord(start_gpu);
    vecadd_gpu(x, y, z, N);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    //////////////////////////////////////

    // Get the elapsed time in milliseconds
    float ms;
    hipEventElapsedTime(&ms, start_gpu, stop_gpu);
    std::cout << "Total GPU time: " << ms << "ms\n";
    // std::cout << z[N-1]<<"\n";

    // free memory
    free(x);
    free(y);
    free(z);

    // stop the profiler
    hipProfilerStop();

    return 0;
}

/*

1. Why is numThreadsPerBlock = 512 ?










*/ 