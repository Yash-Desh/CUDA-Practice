// Author : Yash Deshpande
// Date : 28-12-2024
// Tutor : ECE759, TW. Huang

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void simpleKernel(int *data)
{
    // this adds a value to a variable stored in global memory
    data[threadIdx.x] += 2 * (blockIdx.x + threadIdx.x);
}
int main()
{
    const int numElems = 4;
    int hostArray[numElems], *devArray;
    
    // allocate memory on the device (GPU); zero out all entries in this device array
    hipMalloc((void **)&devArray, sizeof(int) * numElems);
    hipMemset(devArray, 0, numElems * sizeof(int));
    
    // invoke GPU kernel, with one block that has four threads
    simpleKernel<<<1, numElems>>>(devArray);

    // What happens if we invoke the kernel like this:
    // simpleKernel<<<1, 12>>>(devArray);
    // Ans: out-of-bounds access for threadIdx.x >= 4

    // What happens if we invoke the kernel like this:
    // simpleKernel<<<2, 4>>>(devArray);
    // Ans: Race Condition, undefined result
    
    // bring the result back from the GPU into the hostArray
    hipMemcpy(&hostArray, devArray, sizeof(int) * numElems, hipMemcpyDeviceToHost);
    
    // print out the result to confirm that things are looking good
    std::cout << "Values stored in hostArray: " << std::endl;
    for (int i = 0; i < numElems; i++)
        std::cout << hostArray[i] << std::endl;
    // release the memory allocated on the GPU
    hipFree(devArray);
    return 0;
}
