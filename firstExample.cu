#include <hip/hip_runtime.h>
#include <iostream>
__global__ void simpleKernel(int *data)
{
    // this adds a value to a variable stored in global memory
    data[threadIdx.x] += 2 * (blockIdx.x + threadIdx.x);
}
int main()
{
    const int numElems = 4;
    int hostArray[numElems], *devArray;
    // allocate memory on the device (GPU); zero out all entries in this device array
    hipMalloc((void **)&devArray, sizeof(int) * numElems);
    hipMemset(devArray, 0, numElems * sizeof(int));
    // invoke GPU kernel, with one block that has four threads
    simpleKernel<<<1, numElems>>>(devArray);
    // bring the result back from the GPU into the hostArray
    hipMemcpy(&hostArray, devArray, sizeof(int) * numElems, hipMemcpyDeviceToHost);
    // print out the result to confirm that things are looking good
    std::cout << "Values stored in hostArray: " << std::endl;
    for (int i = 0; i < numElems; i++)
        std::cout << hostArray[i] << std::endl;
    // release the memory allocated on the GPU
    hipFree(devArray);
    return 0;
}
